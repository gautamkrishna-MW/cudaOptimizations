#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void reductionKernel(float* inpArr, size_t inpSize, float *outputVal)
{
    size_t gTIdx = blockDim.x * blockIdx.x + threadIdx.x;
    size_t tIdx = threadIdx.x;

    // Shuffle instructions
    size_t warpSize = 32;
    int value = inpArr[gTIdx];
    int num_elements = inpSize;
    
    int num_elements_fullwarp = (warpSize * (num_elements / warpSize));
    int num_elements_rem = num_elements - num_elements_fullwarp;
    
    unsigned int mask_rem = ((1U << num_elements_rem) - 1);
    unsigned int mask = (gTIdx < num_elements_fullwarp) ? -1 : mask_rem;
    
    int tmpVal;
    for (int i = warpSize / 2; i > 0; i = i / 2) {
        tmpVal = __shfl_down_sync(mask, value, i);
        value += tmpVal;
    }
    
    if (tIdx % warpSize == 0)
        atomicAdd(outputVal, value);    
}

#define NUMEL 5000
int main()
{
    //float inputArray[NUMEL] = { 3.48,2.71,-7.64,-3.42,8.05,-2.59,-9.14,6.69,6.11,3.59,-9.32,6.18,2.17, 2.82,-2.7,-2.5,0.88,6.54,-4.06,4.15,-7.62,2.79,4.48,3.62,-0.96,-7.79,-2.49,3.,6.75,-9.65, -4.86,6.,-0.05,4.98,-1.33,9.18,-3.4,3.5,5.08,-7.57,1.28,9.12,-8.78,-4.59,-6.8,5.34,3.62,2.38,3.4,-2.04 };
    /*printf("%d", 110 % 32);
    return 0;*/

    //float inputArray[NUMEL] = { 4,5,2,6,8,9,9,1,8,8,0,3,8,3,8,8,8,6,4,6,4,9,5,6,6,8,4,0,8,7,5,0, 4, 0, 4, 4, 0, 6, 6, 1, 7, 5, 2, 2, 3, 5, 7, 0, 9, 2 };
    float inputArray[NUMEL];
    for (int i = 0; i < NUMEL; i++)
    {
        inputArray[i] = (rand()%100 - 50) / 25.f;
    }

    float sum = 0;
    for (int i = 0; i < NUMEL; i++)
    {
        sum += inputArray[i];
    }
    printf("CPU: %f\n", sum);

    float* devInpArr, *outputVal;
    size_t inpSize = NUMEL;
    hipMalloc((void**)&devInpArr, NUMEL * sizeof(float));
    hipMalloc((void**)&outputVal, sizeof(float));
    hipMemset(outputVal, 0, sizeof(float));
    hipMemcpy(devInpArr, inputArray, NUMEL * sizeof(float), hipMemcpyHostToDevice);

    size_t threads = 512;
    size_t blocks = ((NUMEL / threads) + 1);
    reductionKernel << <blocks, threads >> > (devInpArr, inpSize, outputVal);

    float outValGPU = 0;
    hipMemcpy(&outValGPU, outputVal, sizeof(float), hipMemcpyDeviceToHost);
    printf("GPU: %f, ", outValGPU);

    hipFree(devInpArr);
    hipFree(outputVal);
    return 0;
}
